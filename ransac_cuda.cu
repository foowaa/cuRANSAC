#include "hip/hip_runtime.h"
#include <ransac_cuda.h>
__global__
void ransac_kernel(int** consensus_set_x, int** consensus_set_y,
                    ransac_common_params* common_params,
                    ransac_contour_params* contour_params,
                    ransac_result* result)
{
    int contour_id = blockIdx.x;
    hiprandStateMRG32k3a state;
    hiprand_init(clock(), threadIdx.x, 0, &state);
    int consensus_size = contour_params->consensus_size;
    __shared__ int* votes;
    votes = (int*)(malloc(sizeof(int)*common_params->num_iterations));
    __shared__ ransac_result* iter_result;
    iter_result = (ransac_result*)(malloc(sizeof(ransac_result)*common_params->num_iterations));

    iter_result[threadIdx.x].radius = 0;
    iter_result[threadIdx.x].cy = 0;
    iter_result[threadIdx.x].cx = 0;
    if(threadIdx.x == 0)
    {
        result[contour_id].radius = 0;
        result[contour_id].cx = 0;
        result[contour_id].cy = 0;
    }

    unsigned int iA = hiprand(&state);
    unsigned int iB = hiprand(&state);
    unsigned int iC = hiprand(&state);
    iA %= consensus_size;
    iB %= consensus_size;
    iC %= consensus_size;
    float AB, BC, CA;
    float m_AB, m_BC, b_AB;//, b_BC;
    float x_mp_AB, y_mp_AB, x_mp_BC, y_mp_BC;
    float m_pb_AB, m_pb_BC, b_pb_AB, b_pb_BC;
#if DEBUG == 1
    if(threadIdx.x == 0 )
    printf("A B C : %d %d %d\n", iA, iB, iC);
#endif
    
    AB = norm3df(consensus_set_x[contour_id][iA] - consensus_set_x[contour_id][iB],
                 consensus_set_y[contour_id][iA] - consensus_set_y[contour_id][iB], 0.0);
    BC = norm3df(consensus_set_x[contour_id][iB] - consensus_set_x[contour_id][iC],
                 consensus_set_y[contour_id][iB] - consensus_set_y[contour_id][iC], 0.0);
    CA = norm3df(consensus_set_x[contour_id][iC] - consensus_set_x[contour_id][iA],
                 consensus_set_y[contour_id][iC] - consensus_set_y[contour_id][iA], 0.0);
#if DEBUG == 1
    if(threadIdx.x == 0 && blockIdx.x ==0)
    {
        printf("\nA : %d %d\n", consensus_set_x[contour_id][iA], consensus_set_y[contour_id][iA] );
        printf("B : %d %d\n", consensus_set_x[contour_id][iB], consensus_set_y[contour_id][iB] );
        printf("C : %d %d\n", consensus_set_x[contour_id][iC], consensus_set_y[contour_id][iC] );
        printf("AB BC : %f %f\n", AB, BC );
    }
#endif
    if(AB < common_params->min_point_separation || BC < common_params->min_point_separation ||
       CA < common_params->min_point_separation ||
       AB > common_params->max_point_separation || BC > common_params->max_point_separation ||
       CA > common_params->max_point_separation )
    {
#if DEBUG == 1
        if(threadIdx.x == 0 && blockIdx.x == 0)
        printf("\n\tAB, CA, Max pt separation %f %f %f\n", AB, CA, common_params->max_point_separation);// (float)(consensus_set_x[contour_id][iA] - consensus_set_x[contour_id][iB]), consensus_set_x[contour_id][iC] - consensus_set_x[contour_id][iA], common_params->max_point_separation);
#endif
        return ;
    }

    m_AB = (consensus_set_y[contour_id][iA]-consensus_set_y[contour_id][iB])/(consensus_set_x[contour_id][iA]-consensus_set_x[contour_id][iB] + 0.001);
    m_BC = (consensus_set_y[contour_id][iB]-consensus_set_y[contour_id][iC])/(consensus_set_x[contour_id][iB]-consensus_set_x[contour_id][iC] + 0.001);
    b_AB = (consensus_set_y[contour_id][iB] - m_AB*consensus_set_x[contour_id][iB]);

    if(abs(consensus_set_y[contour_id][iC] - ((m_AB*(consensus_set_x[contour_id][iC])) + b_AB )) < common_params->colinear_tolerance)
    {
        return ;
    }

    x_mp_AB = (consensus_set_x[contour_id][iA]+consensus_set_x[contour_id][iB])/2.0;
    y_mp_AB = (consensus_set_y[contour_id][iA]+consensus_set_y[contour_id][iB])/2.0;
    x_mp_BC = (consensus_set_x[contour_id][iB]+consensus_set_x[contour_id][iC])/2.0;
    y_mp_BC = (consensus_set_y[contour_id][iB]+consensus_set_y[contour_id][iC])/2.0;

    m_pb_AB = -1/m_AB;
    m_pb_BC = -1/m_BC;
    b_pb_AB = y_mp_AB - m_pb_AB*x_mp_AB;
    b_pb_BC = y_mp_BC - m_pb_BC*x_mp_BC;
    
    iter_result[threadIdx.x].cx = (b_pb_AB - b_pb_BC)/(m_pb_BC - m_pb_AB + 0.0001);
    iter_result[threadIdx.x].cy = m_pb_AB*iter_result[threadIdx.x].cx + b_pb_AB;
    iter_result[threadIdx.x].radius = norm3df(iter_result[threadIdx.x].cx - consensus_set_x[contour_id][iA],
                                iter_result[threadIdx.x].cx - consensus_set_y[contour_id][iA], 0.0);
    if(iter_result[threadIdx.x].cx < 0 || iter_result[threadIdx.x].cy < 0)
    {
        printf("AYYOOOOO\n");
        printf("In Thread %d %d %d %f\n", threadIdx.x, iter_result[threadIdx.x].cx,iter_result[threadIdx.x].cy,iter_result[threadIdx.x].radius);
        printf("\nA : %d %d\n", consensus_set_x[contour_id][iA], consensus_set_y[contour_id][iA] );
        printf("B : %d %d\n", consensus_set_x[contour_id][iB], consensus_set_y[contour_id][iB] );
        printf("C : %d %d\n", consensus_set_x[contour_id][iC], consensus_set_y[contour_id][iC] );
        printf("AB BC : %f %f\n", AB, BC );
    }

    votes[threadIdx.x] = 0;
    for (int i = 0; i < contour_params[contour_id].consensus_size; i++)
    {
        if (norm3df(consensus_set_y[contour_id][i] - iter_result[threadIdx.x].cy,
                    consensus_set_x[contour_id][i] - iter_result[threadIdx.x].cx, 0.0) - iter_result[threadIdx.x].radius < common_params->radius_tolerance)
        {
            votes[threadIdx.x]++;
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        int max_votes = 0;
        int max_iter = -1;
        for (int i = 0; i < common_params->num_iterations; i++)
        {
            if (votes[i] > max_votes)
            {
                max_votes = votes[i];
                max_iter = i;
            }
        }
            printf("MAX ITER third %d",  common_params->num_iterations);
        if (max_votes > common_params->points_threshold && max_iter != -1)
        {
            result[contour_id].cx = iter_result[max_iter].cx;
            result[contour_id].cy = iter_result[max_iter].cy;
            result[contour_id].radius = iter_result[max_iter].radius;
        }
        else
        {
            result[contour_id].cx = iter_result[max_iter].cx;
            result[contour_id].cy = iter_result[max_iter].cy;
            result[contour_id].radius = iter_result[max_iter].radius;
        }
    }
}

__host__
void launch_ransac_kernels(int** points_x, int** points_y, ransac_common_params* common_params,
                            ransac_contour_params* contour_params, ransac_result* result)
{
    ransac_common_params* dev_common_params;
    ransac_contour_params* dev_contour_params;
    ransac_result* dev_ransac_result;
    int** dev_points_x;
    int** dev_points_y;
    int** dummy_points_x = (int* *)malloc(sizeof(int*)*common_params->num_contours);
    int** dummy_points_y = (int* *)malloc(sizeof(int*)*common_params->num_contours);
    int i;
    printf("CPU Max Iter : %d\n",  common_params->num_iterations);

    /**
     * Memory Allocations on GPU
     */
    printf("CPU maxptsep : %f\n", common_params->max_point_separation);
    hipMalloc((void **)&dev_common_params, sizeof(ransac_common_params));
    hipMalloc((void **)&dev_contour_params, common_params->num_contours*sizeof(ransac_contour_params));
    hipMalloc((void **)&dev_ransac_result, common_params->num_contours*sizeof(ransac_result));
    
    hipMalloc((void **)&dev_points_x, common_params->num_contours*sizeof(int *));
    hipMalloc((void **)&dev_points_y, common_params->num_contours*sizeof(int *));
    
    for (i = 0; i < common_params->num_contours; i++)
    {
        hipMalloc(&dummy_points_x[i], contour_params[i].consensus_size*sizeof(int));
        hipMalloc(&dummy_points_y[i], contour_params[i].consensus_size*sizeof(int));
    }
    hipMemcpy(dev_points_x, dummy_points_x, common_params->num_contours*sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(dev_points_y, dummy_points_y, common_params->num_contours*sizeof(int *), hipMemcpyHostToDevice);

    /**
     * Memory Copies to device
     */
    hipMemcpy(dev_common_params, common_params, sizeof(ransac_common_params), hipMemcpyHostToDevice);
    hipMemcpy(dev_contour_params, contour_params, common_params->num_contours*sizeof(ransac_contour_params), hipMemcpyHostToDevice);
    for (i = 0; i < common_params->num_contours; i++)
    {
        hipMemcpy(dummy_points_x[i], points_x[i], contour_params[i].consensus_size*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dummy_points_y[i], points_y[i], contour_params[i].consensus_size*sizeof(int), hipMemcpyHostToDevice);
    }
    
    printf("%d %d %f\n", result->cx, result->cy, result->radius);
    if (hipSuccess == hipMemcpy(result, dev_ransac_result, common_params->num_contours*sizeof(ransac_result), hipMemcpyDeviceToHost))
    {
        //printf("AAAAAAAAAAAA\n");
    }
    //printf("%d %d %f\n", result->cx, result->cy, result->radius);
    printf("Number of contours and iter %d %d \n", common_params->num_contours, common_params->num_iterations);
    ransac_kernel<<<common_params->num_contours, common_params->num_iterations>>>(dev_points_x, dev_points_y,
            dev_common_params, dev_contour_params, dev_ransac_result);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(err));
    
//    printf("%d %d %f\n", result->cx, result->cy, result->radius);
    if (hipSuccess == hipMemcpy(result, dev_ransac_result, common_params->num_contours*sizeof(ransac_result), hipMemcpyDeviceToHost))
    {
      //  printf ("BBBBBB\n");
    }
     err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(err));
    printf("Center and Radius %d %d %f", result[0].cx, result[0].cy, result[0].radius);
    
    /**
     * Memory Free on GPU
     */
    for (i = 0; i < common_params->num_contours; i++)
    {
        hipFree(dummy_points_x[i]);
        hipFree(dummy_points_y[i]);
    }
     err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(err));
    free(dummy_points_x);
    free(dummy_points_y);
    hipFree(dev_points_x);
    hipFree(dev_points_y);
    hipFree(dev_common_params);
    hipFree(dev_contour_params);
    hipFree(dev_ransac_result);
     err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(err));

    return;
}
